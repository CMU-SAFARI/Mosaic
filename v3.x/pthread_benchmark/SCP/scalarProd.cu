#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/*
 * This sample calculates scalar products of a 
 * given set of input vector pairs
 */



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>


#include "../benchmark_common.h"
///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProdCPU(
    float *h_C,
    float *h_A,
    float *h_B,
    int vectorN,
    int elementN
);



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cu"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloatSCP(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

//Total number of input vector pairs; arbitrary
const int VECTOR_N = 256;
//Number of elements per vector; arbitrary, 
//but strongly preferred to be a multiple of warp size
//to meet memory coalescing constraints
const int ELEMENT_N = 4096;
//Total number of data elements
const int    DATA_N = VECTOR_N * ELEMENT_N;

const int   DATA_SZ = DATA_N * sizeof(float);
const int RESULT_SZ = VECTOR_N  * sizeof(float);



///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
//int main(int argc, char **argv){
int main_SCP(hipStream_t stream_app, pthread_mutex_t *mutexapp, bool flag){
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
    float *d_A, *d_B, *d_C;
    double delta, ref, sum_delta, sum_ref, L1norm;
    unsigned int hTimer;
    int i;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
      //  cutilDeviceInit(argc, argv);
    //else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data...\n");
        printf("...allocating CPU memory.\n");
        h_A     = (float *)malloc(DATA_SZ);
        h_B     = (float *)malloc(DATA_SZ);
        h_C_CPU = (float *)malloc(RESULT_SZ);
        h_C_GPU = (float *)malloc(RESULT_SZ);

        printf("...allocating GPU memory.\n");
        cutilSafeCall( hipMalloc((void **)&d_A, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_B, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_C, RESULT_SZ) );

        printf("...generating input data in CPU mem.\n");
        srand(123);
        //Generating input data on CPU
        for(i = 0; i < DATA_N; i++){
            h_A[i] = RandFloatSCP(0.0f, 1.0f);
            h_B[i] = RandFloatSCP(0.0f, 1.0f);
        }

        printf("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing 
        cutilSafeCall( hipMemcpyAsync(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice, stream_app) );
        cutilSafeCall( hipMemcpyAsync(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice, stream_app) );
    printf("Data init done.\n");


    printf("Executing GPU kernel...\n");
        //cutilSafeCall( hipDeviceSynchronize() );
		//cutilSafeCall( hipStreamSynchronize(stream_app) );
    	if(flag)
        	cutilSafeCall( hipStreamSynchronize(stream_app) );
    	else
        	cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        scalarProdGPU<<<128, 256, 0, stream_app>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
        cutilCheckMsg("scalarProdGPU() execution failed\n");
       // cutilSafeCall( hipDeviceSynchronize() );

	    pthread_mutex_unlock (mutexapp);
        //cutilSafeCall( hipStreamSynchronize(stream_app) );
	    if(flag)
	        cutilSafeCall( hipStreamSynchronize(stream_app) );
	    else
	        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
    printf("GPU time: %f msecs.\n", cutGetTimerValue(hTimer));

    printf("Reading back GPU result...\n");
        //Read back GPU results to compare them to CPU results
        cutilSafeCall( hipMemcpyAsync(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost,stream_app) );


    printf("Checking GPU results...\n");
        printf("..running CPU scalar product calculation\n");
        scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

        printf("...comparing the results\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        for(i = 0; i < VECTOR_N; i++){
            delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
            ref   = h_C_CPU[i];
            sum_delta += delta;
            sum_ref   += ref;
        }
        L1norm = sum_delta / sum_ref;
    printf("L1 error: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");


    printf("Shutting down...\n");
        cutilSafeCall( hipFree(d_C) );
        cutilSafeCall( hipFree(d_B)   );
        cutilSafeCall( hipFree(d_A)   );
        free(h_C_GPU);
        free(h_C_CPU);
        free(h_B);
        free(h_A);
        cutilCheckError( cutDeleteTimer(hTimer) );

    //hipDeviceReset();
	// I am not sure about that.
    //cutilExit(argc, argv);
	return 0;
}
