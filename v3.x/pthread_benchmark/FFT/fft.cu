#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "parboil.h"
//#include "file.h"

#include "../benchmark_common.h"

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

// Block index
#define  bx  blockIdx.x
#define  by  blockIdx.y
// Thread index
#define tx  threadIdx.x

// Possible values are 2, 4, 8 and 16
#define R 2

inline __device__ float2 operator*( float2 a, float2 b ) { return make_float2( a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x ); }
inline __device__ float2 operator+( float2 a, float2 b ) { return make_float2( a.x + b.x, a.y + b.y ); }
inline __device__ float2 operator-( float2 a, float2 b ) { return make_float2( a.x - b.x, a.y - b.y ); }
inline __device__ float2 operator*( float2 a, float b ) { return make_float2( b*a.x , b*a.y); }

#define COS_PI_8  0.923879533f
#define SIN_PI_8  0.382683432f
#define exp_1_16  make_float2(  COS_PI_8, -SIN_PI_8 )
#define exp_3_16  make_float2(  SIN_PI_8, -COS_PI_8 )
#define exp_5_16  make_float2( -SIN_PI_8, -COS_PI_8 )
#define exp_7_16  make_float2( -COS_PI_8, -SIN_PI_8 )
#define exp_9_16  make_float2( -COS_PI_8,  SIN_PI_8 )
#define exp_1_8   make_float2(  1, -1 )
#define exp_1_4   make_float2(  0, -1 )
#define exp_3_8   make_float2( -1, -1 )
  
__device__ void GPU_FFT2( float2 &v1,float2 &v2 ) { 
  float2 v0 = v1;  
  v1 = v0 + v2; 
  v2 = v0 - v2; 
}

__device__ void GPU_FFT4( float2 &v0,float2 &v1,float2 &v2,float2 &v3) { 
   GPU_FFT2(v0, v2);
   GPU_FFT2(v1, v3);
   v3 = v3 * exp_1_4;
   GPU_FFT2(v0, v1);
   GPU_FFT2(v2, v3);    
}


inline __device__ void GPU_FFT2(float2* v){
  GPU_FFT2(v[0],v[1]);
}

inline __device__ void GPU_FFT4(float2* v){
  GPU_FFT4(v[0],v[1],v[2],v[3] );
}


inline __device__ void GPU_FFT8(float2* v){
  GPU_FFT2(v[0],v[4]);
  GPU_FFT2(v[1],v[5]);
  GPU_FFT2(v[2],v[6]);
  GPU_FFT2(v[3],v[7]);

  v[5]=(v[5]*exp_1_8)*M_SQRT1_2;
  v[6]=v[6]*exp_1_4;
  v[7]=(v[7]*exp_3_8)*M_SQRT1_2;

  GPU_FFT4(v[0],v[1],v[2],v[3]);
  GPU_FFT4(v[4],v[5],v[6],v[7]);
  
}

inline __device__ void GPU_FFT16( float2 *v )
{
    GPU_FFT4( v[0], v[4], v[8], v[12] );
    GPU_FFT4( v[1], v[5], v[9], v[13] );
    GPU_FFT4( v[2], v[6], v[10], v[14] );
    GPU_FFT4( v[3], v[7], v[11], v[15] );

    v[5]  = (v[5]  * exp_1_8 ) * M_SQRT1_2;
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8 ) * M_SQRT1_2;
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8 ) * M_SQRT1_2;
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8 ) * M_SQRT1_2;
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4( v[0],  v[1],  v[2],  v[3] );
    GPU_FFT4( v[4],  v[5],  v[6],  v[7] );
    GPU_FFT4( v[8],  v[9],  v[10], v[11] );
    GPU_FFT4( v[12], v[13], v[14], v[15] );
}
     
__device__ int GPU_expand(int idxL, int N1, int N2 ){ 
  return (idxL/N1)*N1*N2 + (idxL%N1); 
}      

__device__ void GPU_FftIteration(int j, int Ns, float2* data0, float2* data1, int N){ 
  float2 v[R];  	
  int idxS = j;       
  float angle = -2*M_PI*(j%Ns)/(Ns*R);      

  for( int r=0; r<R; r++ ) { 
    v[r] = data0[idxS+r*N/R]; 
    v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle)); 
  }       

#if R == 2 
  GPU_FFT2( v ); 
#endif

#if R == 4
  GPU_FFT4( v );
#endif	 	

#if R == 8
  GPU_FFT8( v );
#endif

#if R == 16
  GPU_FFT16( v );
#endif	 	

  int idxD = GPU_expand(j,Ns,R); 

  for( int r=0; r<R; r++ ){
    data1[idxD+r*Ns] = v[r];	
  } 	

}      

__global__ void GPU_FFT_Global(int Ns, float2* data0, float2* data1, int N) { 
  data0+=bx*N;
  data1+=bx*N;	 
  GPU_FftIteration( tx, Ns, data0, data1, N);  
}      

//int main( int argc, char **argv )
int main_fft(hipStream_t stream_app, pthread_mutex_t *mutexapp, bool flag)
{	
  int n_bytes; 
  int N, B;
  struct pb_TimerSet timers;
  //struct pb_Parameters *params;
  /*params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
    }

  int err = 0;*/
  N= 256;
  B= 1024;
	 
  /*if(argc != 3)
    err |= 1;
  else {
    char* numend;
    N = strtol(argv[1], &numend, 10);
    if(numend == argv[1])
      err |= 2;
    B = strtol(argv[2], &numend, 10);
    if(numend == argv[2])
      err |= 4;
  }

  if(err)
  {
    fprintf(stderr, "Expecting two integers for N and B\n");
    exit(-1);
  }*/

  n_bytes = N*B*sizeof(float2);
    
  pb_InitializeTimerSet(&timers);
  pb_SwitchToTimer(&timers, pb_TimerID_IO);

  float2 *source    = (float2 *)malloc( n_bytes );
  float2 *result    = (float2 *)malloc( n_bytes );
  char *file = (char *) "FFT/array.bin";

  //inputData(file,(float*)source,N*B*2);
  FILE* fid = fopen(file, "r");

  if (fid == NULL)
    {
      fprintf(stderr, "Cannot open input file\n");
      exit(-1);
    }
  fread ((float*)source, sizeof (float), N*B*2, fid);
  fclose (fid); 
  
  // allocate device memory
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
  float2 *d_source, *d_work;
  hipMalloc((void**) &d_source, n_bytes);
  CUERR;
  // copy host memory to device
  hipMemcpyAsync(d_source, source, n_bytes,hipMemcpyHostToDevice, stream_app);
  CUERR;
  hipMalloc((void**) &d_work, n_bytes);
  CUERR;
  hipMemset(d_work, 0,n_bytes);
  CUERR;

  pb_SwitchToTimer(&timers, pb_TimerID_GPU);

    for( int Ns=1; Ns<N; Ns*=R){
      GPU_FFT_Global<<<dim3(B), dim3(N/R),0,stream_app>>>(Ns, d_source, d_work, N);
      float2 *tmp = d_source;
      d_source = d_work;
      d_work = tmp;
    }
	printf("I am out from fft kernel launch\n");
	pthread_mutex_unlock (mutexapp);
	if(flag)
		cutilSafeCall( hipStreamSynchronize(stream_app) );
	
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    // copy device memory to host
    hipMemcpyAsync(result, d_source, n_bytes,hipMemcpyDeviceToHost, stream_app);
    CUERR;
	
	if(flag)
		cutilSafeCall( hipStreamSynchronize(stream_app) );

  hipFree(d_source);
  CUERR;
  hipFree(d_work);
  CUERR;	

  /*if (params->outFile)
    {
      Write result to file 
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      outputData(params->outFile, (float*)result, N*B*2);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }*/

  free(source);
  free(result);
  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  return 0;
}

