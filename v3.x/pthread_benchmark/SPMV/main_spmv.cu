/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

//#include "parboil.h"
#include <stdio.h>
#include <stdlib.h>

#include "../benchmark_common.h"
#include "file.h"
#include "gpu_info.h"
#include "jds_kernels.cu"
#include "spmv_jds.h"

static int generate_vector(float* x_vector, int dim) {
  srand(54321);
  for (int i = 0; i < dim; i++) {
    x_vector[i] = (rand() / (float)RAND_MAX);
  }
  return 0;
}

int main_spmv(hipStream_t stream_app, pthread_mutex_t* mutexapp, bool flag) {
  printf("CUDA accelerated sparse matrix vector multiplication****\n");
  // pb_InitializeTimerSet(&timers);
  // pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  // parameters declaration
  int len;
  int depth;
  int dim;
  int pad = 32;
  int nzcnt_len;

  // host memory allocation
  // matrix
  float* h_data;
  int* h_indices;
  int* h_ptr;
  int* h_perm;
  int* h_nzcnt;
  // vector
  float* h_Ax_vector;
  float* h_x_vector;

  // device memory allocation
  // matrix
  float* d_data;
  int* d_indices;
  int* d_ptr;
  int* d_perm;
  int* d_nzcnt;
  // vector
  float* d_Ax_vector;
  float* d_x_vector;

  // load matrix from files
  // pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData("SPMV/Dubcova2.mtx.bin", &len, &depth, &dim, &nzcnt_len, &pad,
            &h_data, &h_indices, &h_ptr, &h_perm, &h_nzcnt);

  // pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  h_Ax_vector = (float*)malloc(sizeof(float) * dim);
  h_x_vector = (float*)malloc(sizeof(float) * dim);
  generate_vector(h_x_vector, dim);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  // pb_SwitchToTimer(&timers, pb_TimerID_COPY);
  // memory allocation
  hipMalloc((void**)&d_data, len * sizeof(float));
  hipMalloc((void**)&d_indices, len * sizeof(int));
  hipMalloc((void**)&d_ptr, depth * sizeof(int));
  hipMalloc((void**)&d_perm, dim * sizeof(int));
  hipMalloc((void**)&d_nzcnt, nzcnt_len * sizeof(int));
  hipMalloc((void**)&d_x_vector, dim * sizeof(float));
  hipMalloc((void**)&d_Ax_vector, dim * sizeof(float));
  hipMemset((void*)d_Ax_vector, 0, dim * sizeof(float));

  // memory copy
  hipMemcpyAsync(d_data, h_data, len * sizeof(float), hipMemcpyHostToDevice,
                  stream_app);
  hipMemcpyAsync(d_indices, h_indices, len * sizeof(int),
                  hipMemcpyHostToDevice, stream_app);
  hipMemcpyAsync(d_perm, h_perm, dim * sizeof(int), hipMemcpyHostToDevice,
                  stream_app);
  hipMemcpyAsync(d_x_vector, h_x_vector, dim * sizeof(int),
                  hipMemcpyHostToDevice, stream_app);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(jds_ptr_int), h_ptr, depth * sizeof(int), 0,
                          hipMemcpyHostToDevice, stream_app);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(sh_zcnt_int), h_nzcnt, nzcnt_len * sizeof(int), 0,
                          hipMemcpyHostToDevice, stream_app);

  // pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  dim3 grid;
  dim3 block;
  compute_active_thread(&(block.x), &(grid.x), nzcnt_len, pad, deviceProp.major,
                        deviceProp.minor, deviceProp.warpSize,
                        deviceProp.multiProcessorCount);
  //	grid.x=nzcnt_len;
  //	block.x=pad;
  grid.y = 1;
  grid.z = 1;
  block.y = 1;
  block.z = 1;

  // main execution
  // pb_SwitchToTimer(&timers, pb_TimerID_GPU);
  hipBindTexture(0, tex_x_float, d_x_vector);
  spmv_jds_texture<<<grid, block, 0, stream_app>>>(
      d_Ax_vector, d_data, d_indices, d_perm, d_x_vector, d_nzcnt, dim);

  CUERR  // check and clear any existing errors

      hipUnbindTexture(tex_x_float);
  pthread_mutex_unlock(mutexapp);
  if (flag)
    cutilSafeCall(hipStreamSynchronize(stream_app));
  else
    cutilSafeCall(hipDeviceSynchronize());

  // pb_SwitchToTimer(&timers, pb_TimerID_COPY);
  // HtoD memory copy
  hipMemcpyAsync(h_Ax_vector, d_Ax_vector, dim * sizeof(float),
                  hipMemcpyDeviceToHost, stream_app);

  if (flag)
    cutilSafeCall(hipStreamSynchronize(stream_app));

  hipFree(d_data);
  hipFree(d_indices);
  hipFree(d_ptr);
  hipFree(d_perm);
  hipFree(d_nzcnt);
  hipFree(d_x_vector);
  hipFree(d_Ax_vector);

  // pb_SwitchToTimer(&timers, pb_TimerID_IO);
  outputData("SPMV/Dubcova2.mtx.out", h_Ax_vector, dim);

  // pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  free(h_data);
  free(h_indices);
  free(h_ptr);
  free(h_perm);
  free(h_nzcnt);
  free(h_Ax_vector);
  free(h_x_vector);
  // pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  // pb_PrintTimerSet(&timers);

  return 0;
}
