/***********************************************
        streamcluster_cuda_header.cu
        : header file to streamcluster

        - original code from PARSEC Benchmark Suite
        - parallelization with CUDA API has been applied by

        Sang-Ha (a.k.a Shawn) Lee - sl4ge@virginia.edu
        University of Virginia
        Department of Electrical and Computer Engineering
        Department of Computer Science

***********************************************/

#ifndef STREAMCLUSTER_CUDA_HEADER_CU
#define STREAMCLUSTER_CUDA_HEADER_CU
#include <assert.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/resource.h>
#include <sys/time.h>
#include <fstream>
#include <iostream>
#include "../benchmark_common.h"

#include <hip/hip_runtime.h>

#ifdef ENABLE_PARSEC_HOOKS
#include <hooks.h>
#endif

using namespace std;

/* this structure represents a point */
/* these will be passed around to avoid copying coordinates */
typedef struct {
  float weight;
  float* coord;
  long assign; /* number of point where this one is assigned */
  float cost;  /* cost of that assignment, weight*distance */
} Point;

/* this is the array of points */
typedef struct {
  long num; /* number of points; may not be N if this is a sample */
  int dim;  /* dimensionality */
  Point* p; /* the array itself */
} Points;

struct pkmedian_arg_t {
  Points* points;
  long kmin;
  long kmax;
  long* kfinal;
  int pid;
  pthread_barrier_t* barrier;
  hipStream_t stream_app;
  pthread_mutex_t* mutexapp;
  bool flag;
};

class PStream {
 public:
  virtual size_t read(float* dest, int dim, int num) = 0;
  virtual int ferror() = 0;
  virtual int feof() = 0;
  virtual ~PStream() {}
};

// synthetic stream
class SimStream : public PStream {
 public:
  SimStream(long n_) { n = n_; }
  size_t read(float* dest, int dim, int num) {
    size_t count = 0;
    for (int i = 0; i < num && n > 0; i++) {
      for (int k = 0; k < dim; k++) {
        dest[i * dim + k] = lrand48() / (float)INT_MAX;
      }
      n--;
      count++;
    }
    return count;
  }
  int ferror() { return 0; }
  int feof() { return n <= 0; }
  ~SimStream() {}

 private:
  long n;
};

class FileStream : public PStream {
 public:
  FileStream(char* filename) {
    fp = fopen(filename, "rb");
    if (fp == NULL) {
      fprintf(stderr, "error opening file %s\n.", filename);
      exit(1);
    }
  }
  size_t read(float* dest, int dim, int num) {
    return std::fread(dest, sizeof(float) * dim, num, fp);
  }
  int ferror() { return std::ferror(fp); }
  int feof() { return std::feof(fp); }
  ~FileStream() {
    printf("closing file stream\n");
    fclose(fp);
  }

 private:
  FILE* fp;
};

/* function prototypes */
double gettime();
int isIdentical(float*, float*, int);
// static int floatcomp(const void*, const void*);
void shuffle(Points*);
void intshuffle(int*, int);
float waste(float);
float dist(Point, Point, int);
float pspeedy(Points*, float, long, int, pthread_barrier_t*);
float pgain_old(long, Points*, float, long int*, int, pthread_barrier_t*);
float pFL(Points*,
          int*,
          int,
          float,
          long*,
          float,
          long,
          float,
          int,
          pthread_barrier_t*,
          hipStream_t,
          pthread_mutex_t*,
          bool);
int selectfeasible_fast(Points*, int**, int, int, pthread_barrier_t*);
float pkmedian(Points*,
               long,
               long,
               long*,
               int,
               pthread_barrier_t*,
               hipStream_t,
               pthread_mutex_t*,
               bool);
int contcenters(Points*);
void copycenters(Points*, Points*, long*, long);
void* localSearchSub(void*);
void localSearch(Points*,
                 long,
                 long,
                 long*,
                 hipStream_t,
                 pthread_mutex_t*,
                 bool);
void outcenterIDs(Points*, long*, char*);
void streamCluster(PStream*,
                   long,
                   long,
                   int,
                   long,
                   long,
                   char*,
                   hipStream_t,
                   pthread_mutex_t*,
                   bool);
float pgain(long,
            Points*,
            float,
            long int*,
            int,
            bool*,
            int*,
            bool*,
            double*,
            double*,
            double*,
            double*,
            double*,
            hipStream_t,
            pthread_mutex_t*,
            bool);
void allocDevMem(int, int, int);
void freeDevMem();
void quit(char*);

#endif
